
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <cstdlib>

// CUDA Kernel to compute dot product
__global__ void dotProduct(double *a, double *b, double *partialSums, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ double sharedMemory[256]; // Shared memory for partial dot products

    sharedMemory[threadIdx.x] = 0; // Initialize shared memory

    // Compute partial dot products in parallel
    for (int i = index; i < n; i += stride) {
        sharedMemory[threadIdx.x] += a[i] * b[i];
    }

    __syncthreads(); // Ensure all threads have finished updating shared memory

    // Reduction in shared memory to compute partial sums
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            sharedMemory[threadIdx.x] += sharedMemory[threadIdx.x + i];
        }
        __syncthreads(); // Synchronize threads after each reduction step
    }

    // Write the partial sum to global memory
    if (threadIdx.x == 0) {
        partialSums[blockIdx.x] = sharedMemory[0];
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <numBlocks> <threadsPerBlock>" << std::endl;
        return 1;
    }

    int numBlocks = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);

    int n = 1 << 18; // Length of vectors 'a' and 'b'
    std::vector<double> vectorA(n); // Initialize vector 'a' with random values
    std::vector<double> vectorB(n); // Initialize vector 'b' with random values

    // Fill vectors with random values
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(1.0, 10.0); // Generate random values between 1 and 10
    for (int i = 0; i < n; ++i) {
        vectorA[i] = dis(gen);
        vectorB[i] = dis(gen);
    }

    double *deviceA, *deviceB, *devicePartialSums; // Device pointers
    double *partialSums = new double[numBlocks]; // Array to store partial sums on the host
    double result = 0.0; // Variable to store the dot product result

    hipError_t cudaStatus; // Variable to store CUDA function return status

    // Allocate device memory for vectors 'a', 'b', and partial sums
    cudaStatus = hipMalloc(&deviceA, n * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for deviceA!" << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&deviceB, n * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for deviceB!" << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&devicePartialSums, numBlocks * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for devicePartialSums!" << std::endl;
        return 1;
    }

    // Copy input data from host to device memory
    hipMemcpy(deviceA, vectorA.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, vectorB.data(), n * sizeof(double), hipMemcpyHostToDevice);

    // Benchmark CUDA implementation
    auto start = std::chrono::high_resolution_clock::now();
    dotProduct<<<numBlocks, threadsPerBlock>>>(deviceA, deviceB, devicePartialSums, n);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching dotProduct kernel!" << std::endl;
        return 1;
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Num Blocks: " << numBlocks << ", Threads Per Block: " << threadsPerBlock << ", CUDA time: " << duration.count() << " seconds" << std::endl;

    // Copy partial sums from device to host
    hipMemcpy(partialSums, devicePartialSums, numBlocks * sizeof(double), hipMemcpyDeviceToHost);

    // Compute final dot product on the host
    for (int i = 0; i < numBlocks; ++i) {
        result += partialSums[i];
    }

    // Print final dot product
    std::cout << "CUDA dot product: " << result << std::endl;

    // Free device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(devicePartialSums);
    delete[] partialSums;

    return 0;
}
